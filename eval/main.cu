#include "hip/hip_runtime.h"
// Copyright 2022 Ryohei Kobayashi
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// https://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <fcntl.h>
#include <omp.h>
#include <sys/stat.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <unistd.h>

#include <random>
#include <vector>

#ifndef CL_HPP_ENABLE_EXCEPTIONS
#define CL_HPP_ENABLE_EXCEPTIONS
#endif
#ifndef CL_TARGET_OPENCL_VERSION
#define CL_TARGET_OPENCL_VERSION 200
#endif
#ifndef CL_HPP_TARGET_OPENCL_VERSION
#define CL_HPP_TARGET_OPENCL_VERSION 200
#endif
#ifndef CL_USE_DEPRECATED_OPENCL_1_2_APIS
#define CL_USE_DEPRECATED_OPENCL_1_2_APIS
#endif

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wignored-qualifiers"
#pragma GCC diagnostic ignored "-Wunused-function"
#pragma GCC diagnostic ignored "-Wunused-parameter"
#pragma GCC diagnostic ignored "-Wsign-compare"
#include <CL/cl.h>
#include <CL/cl_ext_intelfpga.h>

#include <CL/cl2.hpp>
#pragma GCC diagnostic pop

#include <time.h>

class Timer {
 public:
  Timer();
  ~Timer();
  void Reset();
  void Start();
  void Stop();
  void Display();
  double Seconds();

 private:
  double seconds_;
  double ref_;
};

Timer::Timer() {
  seconds_ = 0.0;
  ref_ = 0.0;
  struct timespec ts;
  clock_getres(CLOCK_MONOTONIC, &ts);
  fprintf(stderr, "Timer Initialized (precision: %ld.%09ld sec)\n",
          (long)ts.tv_sec, ts.tv_nsec);
}

Timer::~Timer() {}

void Timer::Reset() {
  seconds_ = 0.0;
  ref_ = 0.0;
}

void Timer::Start() {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  ref_ = (double)(ts.tv_sec) + (double)ts.tv_nsec * 1e-9;
}

void Timer::Stop() {
  seconds_ -= ref_;
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  ref_ = (double)(ts.tv_sec) + (double)ts.tv_nsec * 1e-9;
  seconds_ += ref_;
}

void Timer::Display() {
  fprintf(stderr, "Elapsed time: \t%lf sec\n", seconds_);
}

double Timer::Seconds() { return seconds_; }

#define CUCHECK(call)                                                         \
  {                                                                           \
    hipError_t err = call;                                                     \
    if (hipSuccess != err) {                                                 \
      fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, \
              __LINE__, hipGetErrorString(err));                             \
      exit(EXIT_FAILURE);                                                     \
    }                                                                         \
  }

struct Elem {
  unsigned int key;
  unsigned int val;
};

struct Arr {
  unsigned int *key;
  unsigned int *val;

  Arr(size_t numdata_) {
    size_t const buf_size = sizeof(unsigned int) * numdata_;
    CUCHECK(hipMalloc(&key, buf_size));
    CUCHECK(hipMalloc(&val, buf_size));
  }

  ~Arr() {
    CUCHECK(hipFree(key));
    CUCHECK(hipFree(val));
  }
};

template <typename T>
class dev_cmp_custom_key {
 public:
  __host__ __device__ bool operator()(const T &lhs, const T &rhs) const {
    return (lhs.key < rhs.key);
  }
};

int main(int argc, char **argv) {
  if (argc == 1) {
    fprintf(stderr,
            "Usage: ./test_fpga_sort.exe <AOCX file> <numdata in log scale>\n");
    // fprintf(stderr, "\n");
    exit(EXIT_FAILURE);
  }
  if (argc != 3) {
    fprintf(stderr, "Error!\nThe number of arguments is wrong.\n");
    exit(EXIT_FAILURE);
  }

  ///// Create platform //////
  cl::Platform platform;
  std::vector<cl::Platform> platforms;
  cl::Platform::get(&platforms);

  for (auto &p : platforms) {
    auto const name = p.getInfo<CL_PLATFORM_NAME>();
    if (name.find("Intel(R) FPGA SDK for OpenCL(TM)") != std::string::npos) {
      platform = p;
      break;
    }
  }

  if (!platform()) {
    throw cl::Error(CL_DEVICE_NOT_FOUND, "Platform not found");
  }
  if (cl::Platform::setDefault(platform) != platform) {
    throw cl::Error(CL_DEVICE_NOT_FOUND, "Platform not found");
  }

  ///// Check aocx //////
  auto const fd = open(argv[1], O_RDONLY);
  if (fd == -1) {
    perror("open");
    throw cl::Error(CL_INVALID_PROGRAM_EXECUTABLE, "open(2)");
  }

  struct stat st;

  if (fstat(fd, &st)) {
    throw cl::Error(CL_INVALID_PROGRAM_EXECUTABLE, "fstat(2)");
  }

  auto data = new char[st.st_size];
  if (read(fd, data, st.st_size) != st.st_size) {
    throw cl::Error(CL_INVALID_PROGRAM_EXECUTABLE, "read(2)");
  }

  ///// Create context //////
  int dev_idx = 0;
  char const *str;
  str = getenv("OMPI_COMM_WORLD_LOCAL_RANK");
  if (str) {
    dev_idx = atoi(str);
  }

  std::vector<cl::Device> devs;
  platform.getDevices(CL_DEVICE_TYPE_ALL, &devs);
  auto const &dev = devs.at(dev_idx);
  auto ctx = cl::Context{dev};

  cl::Context::setDefault(ctx);
  cl::Device::setDefault(dev);

  ///// Create program /////
  auto dev_cl = dev();
  auto len = static_cast<size_t>(st.st_size);
  auto image = (const unsigned char *)data;
  cl_int error;

  auto prg = clCreateProgramWithBinary(ctx(), 1, &dev_cl, &len, &image, nullptr,
                                       &error);
  cl::detail::errHandler(error, "clCreateProgramWithBinary");

  ///// Create command queue /////
  cl::CommandQueue cq0(ctx, dev);

  ///// Create kernel /////
  cl::Kernel k_fpga_sort(cl::Program(prg, true), "intel_fpga_sort");

  ///// Create kernel functor /////
  cl::KernelFunctor<cl::Buffer, cl::Buffer, cl::Buffer, cl_uint, cl::Buffer>
      f_fpga_sort(k_fpga_sort);

  ///// Create buffer (for host and device) /////
  size_t const numdata = (1 << (std::stoull(std::string(argv[2]))));
  size_t const BUF_SIZE = sizeof(cl_uint2) * numdata;

  ///// host buffers
  cl_uint2 *h_send;
  cl_uint2 *h_recv;
  posix_memalign((void **)&h_send, 64, BUF_SIZE);
  posix_memalign((void **)&h_recv, 64, BUF_SIZE);

  ///// device buffers
  cl::Buffer d_dummy(ctx, CL_MEM_READ_WRITE, sizeof(cl_uint2));
  cl::Buffer d_dst(ctx, CL_MEM_READ_WRITE, BUF_SIZE);
  cl::Buffer d_src(ctx, CL_MEM_READ_WRITE, BUF_SIZE);
  cl::Buffer d_ret(ctx, CL_MEM_READ_WRITE, sizeof(cl_ulong));

  ///// Set init data /////
#pragma omp parallel
  {
    std::mt19937 g(omp_get_thread_num() + 1);
    std::uniform_int_distribution<> d(0, INT32_MAX - 1);
#pragma omp for
    for (size_t i = 0; i < numdata; ++i) {
      h_send[i].s[0] = d(g);        // key
      h_send[i].s[1] = 0xffffffff;  // value
    }
  }

  memset(h_recv, 0x0, BUF_SIZE);

  ///// OpenCL-enabled FPGA sort //////
  /////////////////////////////////////
  /// Show this experiment setup
  fprintf(stderr, "Configuration\n");
  fprintf(stderr, "========================\n");
  fprintf(stderr, "numdata = %zu (%zu bytes)\n", numdata, BUF_SIZE);
  fprintf(stderr, "OpenMP Version %d\n", _OPENMP);
  fprintf(stderr, "GPU programming: CUDA Version %d\n", __CUDACC_VER_MAJOR__);
  fprintf(stderr, "FPGA programming: %s\n",
          platform.getInfo<CL_PLATFORM_VERSION>().c_str());
  Timer stop_watch;

  /// Set FPGA data to be sorted
  cq0.enqueueWriteBuffer(d_src, CL_TRUE, 0, BUF_SIZE, h_send);

  /// Invoke FPGA sort
  stop_watch.Reset();
  stop_watch.Start();
  f_fpga_sort(cl::EnqueueArgs(cq0, cl::NDRange(1), cl::NDRange(1)), d_dummy,
              d_dst, d_src, (cl_uint)numdata, d_ret);
  cq0.finish();
  stop_watch.Stop();
  const double fpga_elapsed_time = stop_watch.Seconds();

  /// Do verification
  fprintf(stderr, "\n");
  fprintf(stderr, "Verification\n");
  fprintf(stderr, "========================\n");

  // Retrieve data to be verified from FPGA
  cl_ulong h_ret;
  cq0.enqueueReadBuffer(d_ret, CL_TRUE, 0, sizeof(cl_ulong), &h_ret);
  cq0.enqueueReadBuffer(((h_ret == 0) ? d_src : d_dst), CL_TRUE, 0, BUF_SIZE,
                        h_recv);

  // Prepare correct data by using GPU
  /********** Data structure: AoS **********/
  Elem *gpu_h_buf;
  CUCHECK(hipHostMalloc(&gpu_h_buf, BUF_SIZE));

#pragma omp parallel for
  for (size_t i = 0; i < numdata; ++i) {
    gpu_h_buf[i].key = h_send[i].s[0];
    gpu_h_buf[i].val = h_send[i].s[1];
  }

  Elem *gpu_d_buf;
  CUCHECK(hipMalloc(&gpu_d_buf, BUF_SIZE));

  CUCHECK(hipMemcpy(gpu_d_buf, gpu_h_buf, BUF_SIZE, hipMemcpyHostToDevice));
  thrust::device_ptr<Elem> d_buf_ptr(gpu_d_buf);

  stop_watch.Reset();
  stop_watch.Start();
  thrust::sort(&d_buf_ptr[0], &d_buf_ptr[numdata], dev_cmp_custom_key<Elem>());
  stop_watch.Stop();
  const double gpu_elapsed_time_aos = stop_watch.Seconds();

  CUCHECK(hipMemcpy(gpu_h_buf, gpu_d_buf, BUF_SIZE, hipMemcpyDeviceToHost));

  // Check data
#pragma omp parallel for
  for (int i = 0; i < (int)numdata; i++) {
    if (gpu_h_buf[i].key != h_recv[i].s[0]) {
      fprintf(stderr, "Failed!\n");
      fprintf(stderr, "gpu_h_buf[%d].key = %08x, h_recv[%d].s[0] = %08x\n", i,
              gpu_h_buf[i].key, i, h_recv[i].s[0]);
      exit(EXIT_FAILURE);
    }
  }

  CUCHECK(hipHostFree(gpu_h_buf));
  CUCHECK(hipFree(gpu_d_buf));

  /********** Data structure: SoA **********/
  unsigned int *key_array;
  unsigned int *val_array;
  size_t const buf_size = sizeof(unsigned int) * numdata;
  CUCHECK(hipHostMalloc(&key_array, buf_size));
  CUCHECK(hipHostMalloc(&val_array, buf_size));
#pragma omp parallel for
  for (size_t i = 0; i < numdata; ++i) {
    key_array[i] = h_send[i].s[0];
    val_array[i] = h_send[i].s[1];
  }

  Arr gpu_mem(numdata);

  CUCHECK(hipMemcpy(gpu_mem.key, key_array, buf_size, hipMemcpyHostToDevice));
  CUCHECK(hipMemcpy(gpu_mem.val, val_array, buf_size, hipMemcpyHostToDevice));

  thrust::device_ptr<unsigned int> dev_ptr_key(gpu_mem.key);
  thrust::device_ptr<unsigned int> dev_ptr_val(gpu_mem.val);

  stop_watch.Reset();
  stop_watch.Start();
  thrust::sort_by_key(&dev_ptr_key[0], &dev_ptr_key[numdata], &dev_ptr_val[0]);
  stop_watch.Stop();
  const double gpu_elapsed_time_soa = stop_watch.Seconds();

  CUCHECK(hipMemcpy(key_array, gpu_mem.key, buf_size, hipMemcpyDeviceToHost));
  CUCHECK(hipMemcpy(val_array, gpu_mem.val, buf_size, hipMemcpyDeviceToHost));

  // Check data
#pragma omp parallel for
  for (int i = 0; i < (int)numdata; i++) {
    if (key_array[i] != h_recv[i].s[0]) {
      fprintf(stderr, "Failed!\n");
      fprintf(stderr, "key_array[%d] = %08x, h_recv[%d].s[0] = %08x\n", i,
              key_array[i], i, h_recv[i].s[0]);
      exit(EXIT_FAILURE);
    }
  }

  CUCHECK(hipHostFree(key_array));
  CUCHECK(hipHostFree(val_array));

  // Show result
  fprintf(stderr, "Passed!\n");
  fprintf(stderr, "------------------------------\n");
  for (int i = 0; i < 10; i++) {
    fprintf(stderr, "h_recv[%d].s[0] = %08x\n", i, h_recv[i].s[0]);
  }
  fprintf(stderr, ".....\n");
  for (int i = (int)(numdata - 10); i < (int)numdata; i++) {
    fprintf(stderr, "h_recv[%d].s[0] = %08x\n", i, h_recv[i].s[0]);
  }
  fprintf(stderr, "------------------------------\n");

  fprintf(stderr, "FPGA elapsed time:\t%lf sec\n", fpga_elapsed_time);
  fprintf(stderr, "GPU elapsed time (AoS):\t%lf sec\n", gpu_elapsed_time_aos);
  fprintf(stderr, "GPU elapsed time (SoA):\t%lf sec\n", gpu_elapsed_time_soa);

  free(h_send);
  free(h_recv);

  return 0;
}
